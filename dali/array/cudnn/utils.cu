#include "utils.h"
#include "dali/utils/make_message.h"

#include <atomic>
#ifdef DALI_USE_CUDNN
static hipdnnHandle_t handle;
std::atomic<bool> handle_created(false);
// TODO(szymon): this should be stream specific handle I think.
hipdnnHandle_t* get_handle() {
    bool expected = false;
    bool desired  = true;
    if (handle_created.compare_exchange_strong(expected, desired)) {
       hipdnnCreate(&handle);
    }
    return &handle;
}

inline hipdnnDataType_t cudnn_dtype(DType dtype) {
    if (dtype == DTYPE_FLOAT) {
        return HIPDNN_DATA_FLOAT;
    } else if (dtype == DTYPE_DOUBLE) {
        return HIPDNN_DATA_DOUBLE;
    }

}

template<>
DescriptorHolder<hipdnnFilterDescriptor_t>::DescriptorHolder(const Array& array, bool nchw) {
    CUDNN_CHECK_RESULT(hipdnnCreateFilterDescriptor(&descriptor_),
                       "when creating filter descriptor ");
    hipdnnTensorFormat_t tensor_format = nchw ? HIPDNN_TENSOR_NCHW : HIPDNN_TENSOR_NHWC;
    CUDNN_CHECK_RESULT(hipdnnSetFilter4dDescriptor(
        descriptor_, tensor_format, cudnn_dtype(array.dtype()),
        array.shape()[nchw ? 0 : 1], array.shape()[nchw ? 1 : 3],
        array.shape()[nchw ? 2 : 1], array.shape()[nchw ? 3 : 2]),
        "when setting filter descriptor ");
}

template<>
DescriptorHolder<hipdnnTensorDescriptor_t>::DescriptorHolder(const Array& array, bool nchw) {
    CUDNN_CHECK_RESULT(hipdnnCreateTensorDescriptor(&descriptor_),
                       "when creating tensor descriptor ");
    hipdnnTensorFormat_t tensor_format = nchw ? HIPDNN_TENSOR_NCHW : HIPDNN_TENSOR_NHWC;
    CUDNN_CHECK_RESULT(hipdnnSetTensor4dDescriptor(
        descriptor_, tensor_format, cudnn_dtype(array.dtype()),
        array.shape()[nchw ? 0 : 1], array.shape()[nchw ? 1 : 3],
        array.shape()[nchw ? 2 : 1], array.shape()[nchw ? 3 : 2]),
        "when setting tensor descriptor ");
}

template<>
~DescriptorHolder<hipdnnFilterDescriptor_t>::DescriptorHolder() {
    CUDNN_CHECK_RESULT(hipdnnDestroyFilterDescriptor(&descriptor_),
                       "when destroying filter descriptor ");
}

template<>
~DescriptorHolder<hipdnnTensorDescriptor_t>::DescriptorHolder() {
    CUDNN_CHECK_RESULT(hipdnnDestroyTensorDescriptor(&descriptor_),
                       "when destroying tensor descriptor ");
}

template<>
DescriptorHolder<hipdnnConvolutionDescriptor_t>::DescriptorHolder(int padding_h,
                                                                 int padding_w,
                                                                 int stride_h,
                                                                 int stride_w) {
    CUDNN_CHECK_RESULT(hipdnnCreateConvolutionDescriptor(&descriptor_),
                       "when creating convolution descriptor ");
    CUDNN_CHECK_RESULT(hipdnnSetConvolution2dDescriptor(descriptor_,
                /*pad_h=*/padding_h,
                /*pad_w=*/padding_w,
                /*u=*/stride_h,
                /*v=*/stride_w,
                /*upscalex=*/1,
                /*upscaley=*/1,
                HIPDNN_CROSS_CORRELATION // Theano issue author claims its twice as fast:
                                        // https://github.com/Theano/Theano/issues/3632
            ), "when setting convolution descriptor ");
}

template<>
~DescriptorHolder<hipdnnConvolutionDescriptor_t>::DescriptorHolder() {
    CUDNN_CHECK_RESULT(hipdnnDestroyConvolutionDescriptor(&descriptor_),
                       "when destroying convolution descriptor ");
}

template<>
DescriptorHolder<hipdnnPoolingDescriptor_t>::DescriptorHolder(hipdnnPoolingMode_t pooling_mode,
                                                             int window_h,
                                                             int window_w,
                                                             int padding_h,
                                                             int padding_w,
                                                             int stride_h,
                                                             int stride_w) {
    CUDNN_CHECK_RESULT(cudnnCreatePooling2dDescriptor(&descriptor_),
                       "when creating pooling descriptor ");
    CUDNN_CHECK_RESULT(hipdnnSetPooling2dDescriptor(descriptor_,
                pooling_mode,
                HIPDNN_PROPAGATE_NAN,
                /*windowHeight=*/ window_h,
                /*windowWidth=*/  window_w,
                /*pad_h=*/        padding_h,
                /*pad_w=*/        padding_w,
                /*stride_h=*/     stride_h,
                /*stride_w=*/     stride_w
            ), "when setting pooling descriptor ");
}

template<>
~DescriptorHolder<hipdnnPoolingDescriptor_t>::DescriptorHolder() {
    CUDNN_CHECK_RESULT(cudnnDestroyPooling2dDescriptor(&descriptor_),
                       "when destroying pooling descriptor ");
}

#endif
