#include "utils.h"
#include "dali/utils/make_message.h"

#include <atomic>
#ifdef DALI_USE_CUDNN
static hipdnnHandle_t handle;
std::atomic<bool> handle_created(false);
// TODO(szymon): this should be stream specific handle I think.
hipdnnHandle_t* get_handle() {
    bool expected = false;
    bool desired  = true;
    if (handle_created.compare_exchange_strong(expected, desired)) {
       hipdnnCreate(&handle);
    }
    return &handle;
}

inline hipdnnDataType_t cudnn_dtype(DType dtype) {
    if (dtype == DTYPE_FLOAT) {
        return HIPDNN_DATA_FLOAT;
    } else if (dtype == DTYPE_DOUBLE) {
        return HIPDNN_DATA_DOUBLE;
    } else {
        ASSERT2(false, utils::make_message(
            "CuDNN dtypes are only availabe for float and double but got ",
            dtype, "."));
        return HIPDNN_DATA_FLOAT;
    }
}

DescriptorHolder<hipdnnFilterDescriptor_t>::DescriptorHolder(const std::vector<int>& shape, DType dtype, bool nchw) {
    CUDNN_CHECK_RESULT(hipdnnCreateFilterDescriptor(&descriptor_),
                       "when creating filter descriptor ");
    hipdnnTensorFormat_t tensor_format = nchw ? HIPDNN_TENSOR_NCHW : HIPDNN_TENSOR_NHWC;
    int n, c, h, w;
    if (nchw) {
        n = shape[0]; c = shape[1]; h = shape[2]; w = shape[3];
    } else {
        n = shape[0]; c = shape[3]; h = shape[1]; w = shape[2];
    }
    CUDNN_CHECK_RESULT(hipdnnSetFilter4dDescriptor(
        descriptor_, cudnn_dtype(dtype), tensor_format, n, c, h, w),
        "when setting filter descriptor ");
}

DescriptorHolder<hipdnnTensorDescriptor_t>::DescriptorHolder(const std::vector<int>& shape, DType dtype, bool nchw) {
    CUDNN_CHECK_RESULT(hipdnnCreateTensorDescriptor(&descriptor_),
                       "when creating tensor descriptor ");
    hipdnnTensorFormat_t tensor_format = nchw ? HIPDNN_TENSOR_NCHW : HIPDNN_TENSOR_NHWC;
    int n, c, h, w;
    if (nchw) {
        n = shape[0]; c = shape[1]; h = shape[2]; w = shape[3];
    } else {
        n = shape[0]; c = shape[3]; h = shape[1]; w = shape[2];
    }
    CUDNN_CHECK_RESULT(hipdnnSetTensor4dDescriptor(
        descriptor_, tensor_format, cudnn_dtype(dtype), n, c, h, w),
        "when setting tensor descriptor ");
}

DescriptorHolder<hipdnnFilterDescriptor_t>::~DescriptorHolder() {
    CUDNN_CHECK_RESULT(hipdnnDestroyFilterDescriptor(descriptor_),
                       "when destroying filter descriptor ");
}

DescriptorHolder<hipdnnTensorDescriptor_t>::~DescriptorHolder() {
    CUDNN_CHECK_RESULT(hipdnnDestroyTensorDescriptor(descriptor_),
                       "when destroying tensor descriptor ");
}

DescriptorHolder<hipdnnConvolutionDescriptor_t>::DescriptorHolder(DType dtype,
                                                                 int padding_h,
                                                                 int padding_w,
                                                                 int stride_h,
                                                                 int stride_w) {
    CUDNN_CHECK_RESULT(hipdnnCreateConvolutionDescriptor(&descriptor_),
                       "when creating convolution descriptor ");
    CUDNN_CHECK_RESULT(hipdnnSetConvolution2dDescriptor(descriptor_,
                /*pad_h=*/padding_h,
                /*pad_w=*/padding_w,
                /*u=*/stride_h,
                /*v=*/stride_w,
                /*dilation_h=*/1,
                /*dilation_w=*/1,
                HIPDNN_CROSS_CORRELATION,
                cudnn_dtype(dtype)), "when setting convolution descriptor ");
}

DescriptorHolder<hipdnnConvolutionDescriptor_t>::~DescriptorHolder() {
    CUDNN_CHECK_RESULT(hipdnnDestroyConvolutionDescriptor(descriptor_),
                       "when destroying convolution descriptor ");
}

DescriptorHolder<hipdnnPoolingDescriptor_t>::DescriptorHolder(hipdnnPoolingMode_t pooling_mode,
                                                             int window_h,
                                                             int window_w,
                                                             int padding_h,
                                                             int padding_w,
                                                             int stride_h,
                                                             int stride_w) {
    CUDNN_CHECK_RESULT(hipdnnCreatePoolingDescriptor(&descriptor_),
                       "when creating pooling descriptor ");
    CUDNN_CHECK_RESULT(hipdnnSetPooling2dDescriptor(descriptor_,
                pooling_mode,
                HIPDNN_PROPAGATE_NAN,
                /*windowHeight=*/ window_h,
                /*windowWidth=*/  window_w,
                /*pad_h=*/        padding_h,
                /*pad_w=*/        padding_w,
                /*stride_h=*/     stride_h,
                /*stride_w=*/     stride_w
            ), "when setting pooling descriptor ");
}

DescriptorHolder<hipdnnPoolingDescriptor_t>::~DescriptorHolder() {
    CUDNN_CHECK_RESULT(hipdnnDestroyPoolingDescriptor(descriptor_),
                       "when destroying pooling descriptor ");
}

#endif
