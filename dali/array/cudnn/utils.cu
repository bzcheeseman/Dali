#include "utils.h"

#include <atomic>
#ifdef DALI_USE_CUDNN
static hipdnnHandle_t handle;
std::atomic<bool> handle_created(false);
// TODO(szymon): this should be stream specific handle I think.
hipdnnHandle_t* get_handle() {
    bool expected = false;
    bool desired  = true;
    if (handle_created.compare_exchange_strong(expected, desired)) {
       hipdnnCreate(&handle);
    }
    return &handle;
}

template<>
DescriptorHolder<hipdnnFilterDescriptor_t>::DescriptorHolder(const Array& array, bool nchw) {
    CUDNN_CHECK_RESULT(hipdnnCreateFilterDescriptor(&descriptor_),
                       "when creating filter descriptor ");
    hipdnnTensorFormat_t tensor_format = nchw ? HIPDNN_TENSOR_NCHW : HIPDNN_TENSOR_NHWC;
    CUDNN_CHECK_RESULT(hipdnnSetFilter4dDescriptor(
        descriptor_, tensor_format, cudnn_dtype(array.dtype()),
        array.shape()[nchw ? 0 : 1], array.shape()[nchw ? 1 : 3],
        array.shape()[nchw ? 2 : 1], array.shape()[nchw ? 3 : 2]),
        "when setting filter descriptor ");
}

template<>
DescriptorHolder<hipdnnTensorDescriptor_t>::DescriptorHolder(const Array& array, bool nchw) {
    CUDNN_CHECK_RESULT(hipdnnCreateTensorDescriptor(&descriptor_),
                       "when creating tensor descriptor ");
    hipdnnTensorFormat_t tensor_format = nchw ? HIPDNN_TENSOR_NCHW : HIPDNN_TENSOR_NHWC;
    CUDNN_CHECK_RESULT(hipdnnSetTensor4dDescriptor(
        descriptor_, tensor_format, cudnn_dtype(array.dtype()),
        array.shape()[nchw ? 0 : 1], array.shape()[nchw ? 1 : 3],
        array.shape()[nchw ? 2 : 1], array.shape()[nchw ? 3 : 2]),
        "when setting tensor descriptor ");
}

~DescriptorHolder<hipdnnFilterDescriptor_t>::DescriptorHolder() {
    CUDNN_CHECK_RESULT(hipdnnDestroyFilterDescriptor(&descriptor_),
                       "when destroying filter descriptor ");
}

~DescriptorHolder<hipdnnTensorDescriptor_t>::DescriptorHolder() {
    CUDNN_CHECK_RESULT(hipdnnDestroyTensorDescriptor(&descriptor_),
                       "when destroying tensor descriptor ");
}

template<>
DescriptorHolder<hipdnnConvolutionDescriptor_t>::DescriptorHolder(int padding_h,
                                                                 int padding_w,
                                                                 int stride_h,
                                                                 int stride_w) {
    CUDNN_CHECK_RESULT(hipdnnCreateConvolutionDescriptor(&descriptor_),
                       "when creating convolution descriptor ");
    CUDNN_CHECK_RESULT(hipdnnSetConvolution2dDescriptor(descriptor_,
                /*pad_h=*/padding_h,
                /*pad_w=*/padding_w,
                /*u=*/stride_h,
                /*v=*/stride_w,
                /*upscalex=*/1,
                /*upscaley=*/1,
                HIPDNN_CROSS_CORRELATION // Theano issue author claims its twice as fast:
                                        // https://github.com/Theano/Theano/issues/3632
            ), "when setting convolution descriptor ");
}

~DescriptorHolder<hipdnnConvolutionDescriptor_t>::DescriptorHolder() {
    CUDNN_CHECK_RESULT(hipdnnDestroyConvolutionDescriptor(&descriptor_),
                       "when destroying convolution descriptor ");
}


#endif
